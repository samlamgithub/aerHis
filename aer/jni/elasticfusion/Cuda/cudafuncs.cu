#include "hip/hip_runtime.h"
/*
 * This file is part of ElasticFusion.
 *
 * Copyright (C) 2015 Imperial College London
 *
 * The use of the code within this file and all code within files that
 * make up the software that is ElasticFusion is permitted for
 * non-commercial purposes only.  The full terms and conditions that
 * apply to the code within this file are detailed within the LICENSE.txt
 * file and at <http://www.imperial.ac.uk/dyson-robotics-lab/downloads/elastic-fusion/elastic-fusion-license/>
 * unless explicitly stated.  By downloading this file you agree to
 * comply with these terms.
 *
 * If you wish to use any of this code for commercial purposes then
 * please email researchcontracts.engineering@imperial.ac.uk.
 *
 * Software License Agreement (BSD License)
 *
 *  Copyright (c) 2011, Willow Garage, Inc.
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 *  Author: Anatoly Baskeheev, Itseez Ltd, (myname.mysurname@mycompany.com)
 */

#include "cudafuncs.cuh"
#include "convenience.cuh"
#include "operators.cuh"

__global__ void pyrDownGaussKernel (const PtrStepSz<unsigned short> src, PtrStepSz<unsigned short> dst, float sigma_color)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst.cols || y >= dst.rows)
        return;

    const int D = 5;

    int center = src.ptr (2 * y)[2 * x];

    int x_mi = max(0, 2*x - D/2) - 2*x;
    int y_mi = max(0, 2*y - D/2) - 2*y;

    int x_ma = min(src.cols, 2*x -D/2+D) - 2*x;
    int y_ma = min(src.rows, 2*y -D/2+D) - 2*y;

    float sum = 0;
    float wall = 0;

    float weights[] = {0.375f, 0.25f, 0.0625f} ;

    for(int yi = y_mi; yi < y_ma; ++yi)
        for(int xi = x_mi; xi < x_ma; ++xi)
        {
            int val = src.ptr (2*y + yi)[2*x + xi];

            if (abs (val - center) < 3 * sigma_color)
            {
                sum += val * weights[abs(xi)] * weights[abs(yi)];
                wall += weights[abs(xi)] * weights[abs(yi)];
            }
        }


    dst.ptr (y)[x] = static_cast<int>(sum /wall);
}

void pyrDown(const DeviceArray2D<unsigned short> & src, DeviceArray2D<unsigned short> & dst)
{
    dst.create (src.rows () / 2, src.cols () / 2);

    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    const float sigma_color = 30;

    pyrDownGaussKernel<<<grid, block>>>(src, dst, sigma_color);
    cudaSafeCall ( hipGetLastError () );
};

__global__ void computeVmapKernel(const PtrStepSz<unsigned short> depth, PtrStep<float> vmap, float fx_inv, float fy_inv, float cx, float cy, float depthCutoff)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;

    if(u < depth.cols && v < depth.rows)
    {
        float z = depth.ptr (v)[u] / 1000.f; // load and convert: mm -> meters

        if(z != 0 && z < depthCutoff)
        {
            float vx = z * (u - cx) * fx_inv;
            float vy = z * (v - cy) * fy_inv;
            float vz = z;

            vmap.ptr (v                 )[u] = vx;
            vmap.ptr (v + depth.rows    )[u] = vy;
            vmap.ptr (v + depth.rows * 2)[u] = vz;
        }
        else
        {
            vmap.ptr (v)[u] = __int_as_float(0x7fffffff); /*HIP_NAN_F*/
        }
    }
}

void createVMap(const CameraModel& intr, const DeviceArray2D<unsigned short> & depth, DeviceArray2D<float> & vmap, const float depthCutoff)
{
    vmap.create (depth.rows () * 3, depth.cols ());

    dim3 block (32, 8);
    dim3 grid (1, 1, 1);
    grid.x = getGridDim (depth.cols (), block.x);
    grid.y = getGridDim (depth.rows (), block.y);

    float fx = intr.fx, cx = intr.cx;
    float fy = intr.fy, cy = intr.cy;

    computeVmapKernel<<<grid, block>>>(depth, vmap, 1.f / fx, 1.f / fy, cx, cy, depthCutoff);
    cudaSafeCall (hipGetLastError ());
}

__global__ void computeNmapKernel(int rows, int cols, const PtrStep<float> vmap, PtrStep<float> nmap)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;

    if (u >= cols || v >= rows)
        return;

    if (u == cols - 1 || v == rows - 1)
    {
        nmap.ptr (v)[u] = __int_as_float(0x7fffffff); /*HIP_NAN_F*/
        return;
    }

    float3 v00, v01, v10;
    v00.x = vmap.ptr (v  )[u];
    v01.x = vmap.ptr (v  )[u + 1];
    v10.x = vmap.ptr (v + 1)[u];

    if (!isnan (v00.x) && !isnan (v01.x) && !isnan (v10.x))
    {
        v00.y = vmap.ptr (v + rows)[u];
        v01.y = vmap.ptr (v + rows)[u + 1];
        v10.y = vmap.ptr (v + 1 + rows)[u];

        v00.z = vmap.ptr (v + 2 * rows)[u];
        v01.z = vmap.ptr (v + 2 * rows)[u + 1];
        v10.z = vmap.ptr (v + 1 + 2 * rows)[u];

        float3 r = normalized (cross (v01 - v00, v10 - v00));

        nmap.ptr (v       )[u] = r.x;
        nmap.ptr (v + rows)[u] = r.y;
        nmap.ptr (v + 2 * rows)[u] = r.z;
    }
    else
        nmap.ptr (v)[u] = __int_as_float(0x7fffffff); /*HIP_NAN_F*/
}

void createNMap(const DeviceArray2D<float>& vmap, DeviceArray2D<float>& nmap)
{
    nmap.create (vmap.rows (), vmap.cols ());

    int rows = vmap.rows () / 3;
    int cols = vmap.cols ();

    dim3 block (32, 8);
    dim3 grid (1, 1, 1);
    grid.x = getGridDim (cols, block.x);
    grid.y = getGridDim (rows, block.y);

    computeNmapKernel<<<grid, block>>>(rows, cols, vmap, nmap);
    cudaSafeCall (hipGetLastError ());
}

__global__ void tranformMapsKernel(int rows, int cols, const PtrStep<float> vmap_src, const PtrStep<float> nmap_src,
                                   const mat33 Rmat, const float3 tvec, PtrStepSz<float> vmap_dst, PtrStep<float> nmap_dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < cols && y < rows)
    {
        //vertexes
        float3 vsrc, vdst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
        vsrc.x = vmap_src.ptr (y)[x];

        if (!isnan (vsrc.x))
        {
            vsrc.y = vmap_src.ptr (y + rows)[x];
            vsrc.z = vmap_src.ptr (y + 2 * rows)[x];

            vdst = Rmat * vsrc + tvec;

            vmap_dst.ptr (y + rows)[x] = vdst.y;
            vmap_dst.ptr (y + 2 * rows)[x] = vdst.z;
        }

        vmap_dst.ptr (y)[x] = vdst.x;

        //normals
        float3 nsrc, ndst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
        nsrc.x = nmap_src.ptr (y)[x];

        if (!isnan (nsrc.x))
        {
            nsrc.y = nmap_src.ptr (y + rows)[x];
            nsrc.z = nmap_src.ptr (y + 2 * rows)[x];

            ndst = Rmat * nsrc;

            nmap_dst.ptr (y + rows)[x] = ndst.y;
            nmap_dst.ptr (y + 2 * rows)[x] = ndst.z;
        }

        nmap_dst.ptr (y)[x] = ndst.x;
    }
}

void tranformMaps(const DeviceArray2D<float>& vmap_src,
                  const DeviceArray2D<float>& nmap_src,
                  const mat33& Rmat, const float3& tvec,
                  DeviceArray2D<float>& vmap_dst, DeviceArray2D<float>& nmap_dst)
{
    int cols = vmap_src.cols();
    int rows = vmap_src.rows() / 3;

    vmap_dst.create(rows * 3, cols);
    nmap_dst.create(rows * 3, cols);

    dim3 block(32, 8);
    dim3 grid(1, 1, 1);
    grid.x = getGridDim(cols, block.x);
    grid.y = getGridDim(rows, block.y);

    tranformMapsKernel<<<grid, block>>>(rows, cols, vmap_src, nmap_src, Rmat, tvec, vmap_dst, nmap_dst);
    cudaSafeCall(hipGetLastError());
}

__global__ void copyMapsKernel(int rows, int cols, const float * vmap_src, const float * nmap_src,
                               PtrStepSz<float> vmap_dst, PtrStep<float> nmap_dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < cols && y < rows)
    {
        //vertexes
        float3 vsrc, vdst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

        vsrc.x = vmap_src[y * cols * 4 + (x * 4) + 0];
        vsrc.y = vmap_src[y * cols * 4 + (x * 4) + 1];
        vsrc.z = vmap_src[y * cols * 4 + (x * 4) + 2];

        if(!(vsrc.z == 0))
        {
            vdst = vsrc;
        }

        vmap_dst.ptr (y)[x] = vdst.x;
        vmap_dst.ptr (y + rows)[x] = vdst.y;
        vmap_dst.ptr (y + 2 * rows)[x] = vdst.z;

        //normals
        float3 nsrc, ndst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

        nsrc.x = nmap_src[y * cols * 4 + (x * 4) + 0];
        nsrc.y = nmap_src[y * cols * 4 + (x * 4) + 1];
        nsrc.z = nmap_src[y * cols * 4 + (x * 4) + 2];

        if(!(vsrc.z == 0))
        {
            ndst = nsrc;
        }

        nmap_dst.ptr (y)[x] = ndst.x;
        nmap_dst.ptr (y + rows)[x] = ndst.y;
        nmap_dst.ptr (y + 2 * rows)[x] = ndst.z;
    }
}

void copyMaps(const DeviceArray<float>& vmap_src,
              const DeviceArray<float>& nmap_src,
              DeviceArray2D<float>& vmap_dst,
              DeviceArray2D<float>& nmap_dst)
{
    int cols = vmap_dst.cols();
    int rows = vmap_dst.rows() / 3;

    vmap_dst.create(rows * 3, cols);
    nmap_dst.create(rows * 3, cols);

    dim3 block(32, 8);
    dim3 grid(1, 1, 1);
    grid.x = getGridDim(cols, block.x);
    grid.y = getGridDim(rows, block.y);

    copyMapsKernel<<<grid, block>>>(rows, cols, vmap_src, nmap_src, vmap_dst, nmap_dst);
    cudaSafeCall(hipGetLastError());
}

__global__ void pyrDownKernelGaussF(const PtrStepSz<float> src, PtrStepSz<float> dst, float * gaussKernel)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst.cols || y >= dst.rows)
        return;

    const int D = 5;

    float center = src.ptr (2 * y)[2 * x];

    int tx = min (2 * x - D / 2 + D, src.cols - 1);
    int ty = min (2 * y - D / 2 + D, src.rows - 1);
    int cy = max (0, 2 * y - D / 2);

    float sum = 0;
    int count = 0;

    for (; cy < ty; ++cy)
    {
        for (int cx = max (0, 2 * x - D / 2); cx < tx; ++cx)
        {
            if(!isnan(src.ptr (cy)[cx]))
            {
                sum += src.ptr (cy)[cx] * gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
                count += gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
            }
        }
    }
    dst.ptr (y)[x] = (float)(sum / (float)count);
}

template<bool normalize>
__global__ void resizeMapKernel(int drows, int dcols, int srows, const PtrStep<float> input, PtrStep<float> output)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= dcols || y >= drows)
        return;

    const float qnan = __int_as_float(0x7fffffff);

    int xs = x * 2;
    int ys = y * 2;

    float x00 = input.ptr (ys + 0)[xs + 0];
    float x01 = input.ptr (ys + 0)[xs + 1];
    float x10 = input.ptr (ys + 1)[xs + 0];
    float x11 = input.ptr (ys + 1)[xs + 1];

    if (isnan (x00) || isnan (x01) || isnan (x10) || isnan (x11))
    {
        output.ptr (y)[x] = qnan;
        return;
    }
    else
    {
        float3 n;

        n.x = (x00 + x01 + x10 + x11) / 4;

        float y00 = input.ptr (ys + srows + 0)[xs + 0];
        float y01 = input.ptr (ys + srows + 0)[xs + 1];
        float y10 = input.ptr (ys + srows + 1)[xs + 0];
        float y11 = input.ptr (ys + srows + 1)[xs + 1];

        n.y = (y00 + y01 + y10 + y11) / 4;

        float z00 = input.ptr (ys + 2 * srows + 0)[xs + 0];
        float z01 = input.ptr (ys + 2 * srows + 0)[xs + 1];
        float z10 = input.ptr (ys + 2 * srows + 1)[xs + 0];
        float z11 = input.ptr (ys + 2 * srows + 1)[xs + 1];

        n.z = (z00 + z01 + z10 + z11) / 4;

        if (normalize)
            n = normalized (n);

        output.ptr (y        )[x] = n.x;
        output.ptr (y + drows)[x] = n.y;
        output.ptr (y + 2 * drows)[x] = n.z;
    }
}

template<bool normalize>
void resizeMap(const DeviceArray2D<float>& input, DeviceArray2D<float>& output)
{
    int in_cols = input.cols ();
    int in_rows = input.rows () / 3;

    int out_cols = in_cols / 2;
    int out_rows = in_rows / 2;

    output.create (out_rows * 3, out_cols);

    dim3 block (32, 8);
    dim3 grid (getGridDim (out_cols, block.x), getGridDim (out_rows, block.y));
    resizeMapKernel<normalize><< < grid, block>>>(out_rows, out_cols, in_rows, input, output);
    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall (hipDeviceSynchronize ());
}

void resizeVMap(const DeviceArray2D<float>& input, DeviceArray2D<float>& output)
{
    resizeMap<false>(input, output);
}

void resizeNMap(const DeviceArray2D<float>& input, DeviceArray2D<float>& output)
{
    resizeMap<true>(input, output);
}

void pyrDownGaussF(const DeviceArray2D<float>& src, DeviceArray2D<float> & dst)
{
    dst.create (src.rows () / 2, src.cols () / 2);

    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    const float gaussKernel[25] = {1, 4, 6, 4, 1,
                    4, 16, 24, 16, 4,
                    6, 24, 36, 24, 6,
                    4, 16, 24, 16, 4,
                    1, 4, 6, 4, 1};

    float * gauss_cuda;

    hipMalloc((void**) &gauss_cuda, sizeof(float) * 25);
    hipMemcpy(gauss_cuda, &gaussKernel[0], sizeof(float) * 25, hipMemcpyHostToDevice);

    pyrDownKernelGaussF<<<grid, block>>>(src, dst, gauss_cuda);
    cudaSafeCall ( hipGetLastError () );

    hipFree(gauss_cuda);
};

__global__ void pyrDownKernelIntensityGauss(const PtrStepSz<unsigned char> src, PtrStepSz<unsigned char> dst, float * gaussKernel)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst.cols || y >= dst.rows)
        return;

    const int D = 5;

    int center = src.ptr (2 * y)[2 * x];

    int tx = min (2 * x - D / 2 + D, src.cols - 1);
    int ty = min (2 * y - D / 2 + D, src.rows - 1);
    int cy = max (0, 2 * y - D / 2);

    float sum = 0;
    int count = 0;

    for (; cy < ty; ++cy)
        for (int cx = max (0, 2 * x - D / 2); cx < tx; ++cx)
        {
            //This might not be right, but it stops incomplete model images from making up colors
            if(src.ptr (cy)[cx] > 0)
            {
                sum += src.ptr (cy)[cx] * gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
                count += gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
            }
        }
    dst.ptr (y)[x] = (sum / (float)count);
}

void pyrDownUcharGauss(const DeviceArray2D<unsigned char>& src, DeviceArray2D<unsigned char> & dst)
{
    dst.create (src.rows () / 2, src.cols () / 2);

    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    const float gaussKernel[25] = {1, 4, 6, 4, 1,
                    4, 16, 24, 16, 4,
                    6, 24, 36, 24, 6,
                    4, 16, 24, 16, 4,
                    1, 4, 6, 4, 1};

    float * gauss_cuda;

    hipMalloc((void**) &gauss_cuda, sizeof(float) * 25);
    hipMemcpy(gauss_cuda, &gaussKernel[0], sizeof(float) * 25, hipMemcpyHostToDevice);

    pyrDownKernelIntensityGauss<<<grid, block>>>(src, dst, gauss_cuda);
    cudaSafeCall ( hipGetLastError () );

    hipFree(gauss_cuda);
};

__global__ void verticesToDepthKernel(const float * vmap_src, PtrStepSz<float> dst, float cutOff)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst.cols || y >= dst.rows)
        return;

    float z = vmap_src[y * dst.cols * 4 + (x * 4) + 2];

    dst.ptr(y)[x] = z > cutOff || z <= 0 ? __int_as_float(0x7fffffff)/*HIP_NAN_F*/ : z;
}

void verticesToDepth(DeviceArray<float>& vmap_src, DeviceArray2D<float> & dst, float cutOff)
{
    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    verticesToDepthKernel<<<grid, block>>>(vmap_src, dst, cutOff);
    cudaSafeCall ( hipGetLastError () );
};

texture<uchar4, 2, hipReadModeElementType> inTex;

__global__ void bgr2IntensityKernel(PtrStepSz<unsigned char> dst)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst.cols || y >= dst.rows)
        return;

    uchar4 src = tex2D(inTex, x, y);

    int value = (float)src.x * 0.114f + (float)src.y * 0.299f + (float)src.z * 0.587f;

    dst.ptr (y)[x] = value;
}

void imageBGRToIntensity(hipArray * cuArr, DeviceArray2D<unsigned char> & dst)
{
LOGI("My elasticfusion cuda imageBGRToIntensity 1");
    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));
LOGI("My elasticfusion  cuda imageBGRToIntensity 2");
if (cuArr) {
	LOGI("My elasticfusion cuda imageBGRToIntensity cuArr is not NULL");
} else {
	LOGI("My elasticfusion cuda imageBGRToIntensity cuArr is NULL");
}
if (cuArr == NULL) {
	LOGI("My elasticfusion cuda imageBGRToIntensity cuArr is NULL again ");
} else {
	LOGI("My elasticfusion cuda imageBGRToIntensity cuArr is not NULL again");
}

hipError_t err = hipBindTextureToArray(inTex, cuArr)
if(hipSuccess != err) {
  LOGI("elasticfusion CUDA hipBindTextureToArray error: %s", hipGetErrorString(err));
}
    // cudaSafeCall(hipBindTextureToArray(inTex, cuArr));
LOGI("My elasticfusion cuda imageBGRToIntensity 3");
    bgr2IntensityKernel<<<grid, block>>>(dst);
LOGI("My elasticfusion cuda imageBGRToIntensity 4");
    cudaSafeCall(hipGetLastError());

    cudaSafeCall(hipUnbindTexture(inTex));
    LOGI("My elasticfusion cuda imageBGRToIntensity 5");
};

__constant__ float gsobel_x3x3[9];
__constant__ float gsobel_y3x3[9];

template<int BLOCK_SIZE_X, int BLOCK_SIZE_Y, int PIXELS_PER_THREAD, int N, int N2>
__global__ void sobelKernel(const unsigned char* input_data,
                            unsigned short height,
                            unsigned short width,
                            unsigned short input_pitch,
                            unsigned short output_pitch,
                            short* output_dx,
                            short* output_dy)
{

    short j = (blockIdx.x * BLOCK_SIZE_X) + threadIdx.x;
    short j0 = (blockIdx.x * BLOCK_SIZE_X * PIXELS_PER_THREAD) + (threadIdx.x * PIXELS_PER_THREAD);
    short i = (blockIdx.y * BLOCK_SIZE_Y) + threadIdx.y;

    unsigned int *ptr_ui;
    short *ptr_output_data;

    //Alloc and init shared memory
    __shared__ unsigned int input_data_smem[BLOCK_SIZE_Y + (N << 1)][BLOCK_SIZE_X + (N2 << 1)];
    unsigned char *ptr_smem = (unsigned char*) &(input_data_smem[0][0]);
    unsigned short smem_pitch = (BLOCK_SIZE_X + (N2 << 1)) << 2;

    __shared__ float output_dx_smem[BLOCK_SIZE_Y][BLOCK_SIZE_X * PIXELS_PER_THREAD];
    __shared__ float output_dy_smem[BLOCK_SIZE_Y][BLOCK_SIZE_X * PIXELS_PER_THREAD];

#pragma unroll
    for(short p = 0; p < PIXELS_PER_THREAD; p++)
    {
        output_dx_smem[threadIdx.y][(threadIdx.x * PIXELS_PER_THREAD) + p] = 0;
        output_dy_smem[threadIdx.y][(threadIdx.x * PIXELS_PER_THREAD) + p] = 0;
    }

    if(i < height && j < (width >> 2))
    { //Assume PIXELS_PER_THREAD = 4

        //Each thread loads 1 uint, ie 4 uchar

        //Copy data to shared memory ----------------------------------------------------------------------------

        //1. All threads read, shift up and left
        ptr_ui = (unsigned int*) (input_data + ((i - N) * input_pitch));
        input_data_smem[threadIdx.y][threadIdx.x] = ptr_ui[j - N2];
        //2. Right columns
        if(threadIdx.x < (N2 << 1))
        {
            input_data_smem[threadIdx.y][threadIdx.x + BLOCK_SIZE_X] =
                            (i - N >= 0 && j - N2 + BLOCK_SIZE_X < (width >> 2)) ? ptr_ui[j
                                                                                          - N2 + BLOCK_SIZE_X] : 0;
        }
        //3. Bottom rows
        if(threadIdx.y < (N << 1))
        {
            ptr_ui = (unsigned int*) (input_data
                            + ((i - N + BLOCK_SIZE_Y) * input_pitch));
            input_data_smem[threadIdx.y + BLOCK_SIZE_Y][threadIdx.x] =
                            (i - N + BLOCK_SIZE_Y < height && j - N2 >= 0) ? ptr_ui[j
                                                                                    - N2] : 0;
        }
        //4. Bottom-right
        if(threadIdx.x < (N2 << 1) && threadIdx.y < (N << 1))
        {
            input_data_smem[threadIdx.y + BLOCK_SIZE_Y][threadIdx.x
                                                        + BLOCK_SIZE_X] =
                                                                        (i - N + BLOCK_SIZE_Y < height
                                                                                        && j - N2 + BLOCK_SIZE_X
                                                                                        < (width >> 2)) ? ptr_ui[j
                                                                                                                 - N2 + BLOCK_SIZE_X] : 0;
        }
        __syncthreads();
        //-------------------------------------------------------------------------------------------------------

        //Processing --------------------------------------------------------------------------------------------
        short li = threadIdx.y + N;
        short lj = ((threadIdx.x + N2) * PIXELS_PER_THREAD);

        //3x3 neighbours
        short k = -N, l = -N;
#pragma unroll
        for(short loop = 0; loop < ((N << 1) + 1) * ((N << 1) + 1); loop++)
        {

            short lik = li + k;
            short ljl = lj + l;

            //Get neighbour value
            unsigned char *ptr2 = ptr_smem + (lik * smem_pitch);

            int idx = ((N << 1) + 1) * ((N << 1) + 1) - 1 - loop;
            float factor_x = gsobel_x3x3[idx];
            float factor_y = gsobel_y3x3[idx];

#pragma unroll
            for(short p = 0; p < PIXELS_PER_THREAD; p++)
            {

                //Get current_pixel value
                //  unsigned char val0 = ptr[lj+p];

                float valn = (float) ptr2[ljl + p];
                output_dx_smem[threadIdx.y][(threadIdx.x * PIXELS_PER_THREAD)
                                            + p] += factor_x * valn;
                output_dy_smem[threadIdx.y][(threadIdx.x * PIXELS_PER_THREAD)
                                            + p] += factor_y * valn;

            }                //end for p

            l = (l < N) ? l + 1 : -N;
            k = (l == -N) ? k + 1 : k;
        }                //end loop k,l

        __syncthreads();
        //-------------------------------------------------------------------------------------------------------

#pragma unroll
        for(short p = 0; p < PIXELS_PER_THREAD; p++)
        {
            ptr_output_data = (short*) ((unsigned char *) output_dx + (i * output_pitch));
            ptr_output_data[j0 + p] = (short) output_dx_smem[threadIdx.y][(threadIdx.x * PIXELS_PER_THREAD) + p];
            ptr_output_data = (short*) ((unsigned char *) output_dy + (i * output_pitch));
            ptr_output_data[j0 + p] = (short) output_dy_smem[threadIdx.y][(threadIdx.x * PIXELS_PER_THREAD) + p];
        }
    } //end if
}

void sobelGaussian(DeviceArray2D<unsigned char>& src, DeviceArray2D<short>& dx, DeviceArray2D<short>& dy)
{
    float gsx3x3[9] = {0.52201,  0.00000, -0.52201,
                    0.79451, -0.00000, -0.79451,
                    0.52201,  0.00000, -0.52201};

    float gsy3x3[9] = {0.52201, 0.79451, 0.52201,
                    0.00000, 0.00000, 0.00000,
                    -0.52201, -0.79451, -0.52201};

    hipMemcpyToSymbol(HIP_SYMBOL(gsobel_x3x3), gsx3x3, 9<<2);
    hipMemcpyToSymbol(HIP_SYMBOL(gsobel_y3x3), gsy3x3, 9<<2);

    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall (hipDeviceSynchronize ());

    sobelKernel<32, 6, 4, 1, 1><<<dim3(getGridDim(dx.cols() / 4, 32), getGridDim(dx.rows(), 6)), dim3(32, 6)>>>(src.ptr(0),
                                                                                                                (unsigned short) src.rows(),
                                                                                                                (unsigned short) src.cols(),
                                                                                                                (unsigned short) src.step(),
                                                                                                                (unsigned short) dx.step(),
                                                                                                                dx.ptr(0),
                                                                                                                dy.ptr(0));

    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall (hipDeviceSynchronize ());
}

__global__ void projectPointsKernel(const PtrStepSz<float> depth,
                                    PtrStepSz<float3> cloud,
                                    const float invFx,
                                    const float invFy,
                                    const float cx,
                                    const float cy)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= depth.cols || y >= depth.rows)
        return;

    float z = depth.ptr(y)[x];

    cloud.ptr(y)[x].x = (float)((x - cx) * z * invFx);
    cloud.ptr(y)[x].y = (float)((y - cy) * z * invFy);
    cloud.ptr(y)[x].z = z;
}

void projectToPointCloud(const DeviceArray2D<float> & depth,
                         const DeviceArray2D<float3> & cloud,
                         CameraModel & intrinsics,
                         const int & level)
{
    dim3 block (32, 8);
    dim3 grid (getGridDim (depth.cols (), block.x), getGridDim (depth.rows (), block.y));

    CameraModel intrinsicsLevel = intrinsics(level);

    projectPointsKernel<<<grid, block>>>(depth, cloud, 1.0f / intrinsicsLevel.fx, 1.0f / intrinsicsLevel.fy, intrinsicsLevel.cx, intrinsicsLevel.cy);
    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall (hipDeviceSynchronize ());
}
